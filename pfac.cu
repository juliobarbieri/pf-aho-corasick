
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <string.h>
#include <iostream>

#define BLOCOS     1
//#define THREAD

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)

using namespace std;

typedef struct automato {
	char letra;
	automato *prox;
	automato *ant;
	automato *inf;
	int final;
} Automato;

__global__ void pfac(Automato* at, int *matches, char *frase){

	int x = blockDim.x * blockIdx.x + threadIdx.x;


}

Automato* newAutomato(Automato* ant) {
	Automato *nv = (Automato*) malloc(sizeof(Automato));
	nv->prox = NULL;
	nv->inf = NULL;
	nv->ant = ant;

	return nv;
}

Automato* addAlgarismo(Automato *at, char algm, int first) {
	if (at != NULL && at->letra == algm && first == 1) {
		return at;
	}
	// Caso algarismo novo seja diferente do algarismo da raiz
	else if (at != NULL && at->letra != algm && first == 1) {
		Automato *pt = at->inf;
		Automato *ant = pt;
		while (pt != NULL) {
			if (pt->letra == algm) {
				return pt;
			}
			else {
				if (pt != NULL) {
					ant = pt;
					pt = pt->inf;
				}

			}
		}
		Automato *nv = newAutomato(at);
		nv->letra = algm;
		if (ant != NULL) {
			ant->inf = nv;
			return ant->inf;
		}
		else {
			at->inf = nv;
			return at->inf;
		}

	}

	else if(at != NULL && first == 0)
	{
		Automato *pt = at->prox;
		Automato *ant = NULL;
		while (pt != NULL) {

			if (pt->letra == algm) {
				return pt;
			}
			else
			{
				ant = pt;
				pt = pt->inf;
			}
		}

		Automato *nv = newAutomato(at);
		nv->letra = algm;

		if (ant != NULL) {
			ant->inf = nv;
		}
		else {
			at->prox = nv;
		}

		return nv;
	}
	else
	{
		Automato *nv = newAutomato(NULL);
		nv->letra = algm;
		return nv;
	}
}

void imprimir(Automato *at)
{
	Automato *temp = at;

	while (temp != NULL) {
		printf("%c ", temp->letra);
		imprimir(temp->prox);
		temp = temp->inf;
		printf("\n");
	}

}

/*Automato* mallocGPU(Automato *at)
{
	Automato *temp = at;

	while (temp != NULL) {
		imprimir(temp->prox);
		temp = temp->inf;
	}
}*/

int main (int argc, char **argv)
{
	int GPU = 0;

	Automato *at = newAutomato(NULL);
	at->letra = 'a';
	at->prox = NULL;

	char frase[255] = "ab abg bede ef"; //"abc acd abb agd acc";
	int THREADS = strlen(frase);

	Automato *temp = at;

	int i = 0;
	int first = 1;

	while(frase[i] != '\0')
	{
		if(frase[i] != ' ')
		{
			temp = addAlgarismo(temp, frase[i], first);
			first = 0;
			//printf("Letra: %c\n", temp->letra);
		}
		else
		{
			temp->final = 1;
			temp = at;
			first = 1;
		}
		i++;

	}
	imprimir(at);

	// CPU
	char h_fita[255] = "ab abg bede ef";
	int *h_matches = (int*) malloc(sizeof(int));

	// GPU
	Automato *d_at = NULL;
	char *d_fita = NULL;
	int *d_matches = NULL;

	CHECK_ERROR(hipSetDevice(GPU));

	*h_matches = 0;

	//Reset na GPU selecionada
	CHECK_ERROR(hipDeviceReset());

	CHECK_ERROR(hipMalloc((void**) &d_at, sizeof(Automato*)));
	CHECK_ERROR(hipMalloc((void**) &d_fita, 255*sizeof(char)));
	CHECK_ERROR(hipMalloc((void**) &d_matches, sizeof(int)));

	//Copiando CPU --> GPU
	CHECK_ERROR(hipMemcpy(d_at, at, sizeof(Automato*),  hipMemcpyHostToDevice));
	CHECK_ERROR(hipMemcpy(d_fita, h_fita, 255*sizeof(char),  hipMemcpyHostToDevice));
	CHECK_ERROR(hipMemcpy(d_matches, h_matches, sizeof(int),  hipMemcpyHostToDevice));


	pfac <<<BLOCOS, THREADS>>> (d_at, d_matches, d_fita);

	//Copiando GPU --> CPU
	CHECK_ERROR(hipMemcpy(at, d_at, sizeof(Automato*),  hipMemcpyDeviceToHost));
	CHECK_ERROR(hipMemcpy(h_fita, d_fita, 255*sizeof(char),  hipMemcpyDeviceToHost));
	CHECK_ERROR(hipMemcpy(h_matches, d_matches, sizeof(int),  hipMemcpyDeviceToHost));

	// Liberando memória na GPU
	CHECK_ERROR(hipFree(d_at));
	CHECK_ERROR(hipFree(d_fita));
	CHECK_ERROR(hipFree(d_matches));

	// Liberando memória na CPU
	free(at);
	free(h_matches);
	free(h_fita);


   return EXIT_SUCCESS;
}



