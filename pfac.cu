
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

#define DOMINO 		  4096
#define BLOCOS        8
#define THREAD		  

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)

typedef struct automato {
	char letra;
	automato *prox;
	automato *ant;
	automato *inf;
	int final;
} Automato;

__global__ void automato(char** alfabeto, char *frase){
	int matchs = 0;
	int xAtual = 0;
	int yAtual = 0;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int i = 0;
	for (i = 0; i < 5; i++) {
		if (frase[x] != alfabeto[xAtual][yAtual])
			break;
		else
			matchs++;
	}
	
	
}

Automato* newAutomato(Automato* ant) {
	Automato *nv = (Automato*) malloc(sizeof(Automato));
	nv->prox = NULL;
	nv->inf = NULL;
	nv->ant = ant;
	
	return nv;
}

Automato* addAlgarismo(Automato *at, char algm) {
	if(at != NULL)
	{
		Automato *pt = at->prox;
		Automato *ant = NULL;	
		while (pt != NULL) {
		
			if (pt->letra == algm) {
				return pt;
			}
			else
			{				
				ant = pt;
				pt = pt->inf;
			}
		}
	
		Automato *nv = newAutomato(at);
		nv->letra = algm;
		
		if (ant != NULL) {
			ant->inf = nv;
		}
		else {
			at->prox = nv;
		}
	
		return nv;
	}
	else
	{
		Automato *nv = newAutomato(NULL);
		nv->letra = algm;
		return nv;
	}
}

void imprimir(Automato *at)
{
	char frase[255];
	int i =0;
	int pos = 0;
	Automato *temp = at;	
	while(temp != NULL)
	{	
		Automato *ant;
		do
		{
			frase[i] = temp->letra;			
			i++;
			ant = temp; 
			temp= temp->prox;
		}while(temp != NULL);
		temp = ant;
		
		int j = 0;
		while(j <= pos && temp != NULL )
		{
			temp = temp->inf;
			j++;
		}
		pos++;
		
		if(temp == NULL)
		{
			temp = ant->ant; //Caralho!
			pos = 0;
		}
		printf("Run to the Hills\n");
						
	}
	printf("%s\n",frase);	

}

int main (int argc, char **argv)
{	
	Automato *at = newAutomato(NULL);
	at->letra = 'a';
	at->prox = NULL;
	
	

	//char m[3][3] = {"abc",
	//				"acc",
	//				"adc"};
	
	char frase[255] = "abc acd abb agd acc";
	Automato *temp = at;
	int i=0;
	while(frase[i] != '\0')
	{
		if(frase[i] != ' ')
		{
			temp = addAlgarismo(temp, frase[i]);
			//printf("Letra: %c\n", temp->letra); 
		}
		else
		{
			temp->final = 1;
			temp = at;
		}
		i++;
		
	}  
	imprimir(at);
   return EXIT_SUCCESS;
}





